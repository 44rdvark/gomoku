#include "hip/hip_runtime.h"
#include <cstdio>
#include "constants.h"

extern "C" {

__shared__ char board[EDGE2];
__shared__ int sum;

__device__
int cur(int x, int y) { return x + EDGE * y; }

// VERTICAL
__device__
bool h1(int x, int y, int k) { return y - k >= 0; }
__device__
int m1(int x, int y, int k) { return (x + EDGE * (y - k)); }
__device__
bool h2(int x, int y, int k)  { return y < EDGE - k; }
__device__
int m2(int x, int y, int k) { return (x + EDGE * (y + k)); }

// HORIZONTAL
__device__
bool h3(int x, int y, int k) { return x < EDGE - k; }
__device__
int m3(int x, int y, int k) { return (x + EDGE * y + k); }

__device__
bool h4(int x, int y, int k)  { return x - k >= 0; }
__device__
int m4(int x, int y, int k) { return x + EDGE * y - k; }

// DIAGONAL

__device__
bool h5(int x, int y, int k) { return y - k >= 0 && x - k >= 0; }
__device__
int m5(int x, int y, int k) { return x - k + EDGE * (y - k); }

__device__
bool h6(int x, int y, int k) { return x < EDGE - k && y < EDGE - k; }
__device__
int m6(int x, int y, int k) { return x + k + EDGE * (y + k); }

__device__
bool h7(int x, int y, int k)  { return y < EDGE - k && x - k >= 0; }
__device__
int m7(int x, int y, int k) { return x - k + EDGE * (y + k); }

__device__
bool h8(int x, int y, int k)  { return y - k >= 0 && x < EDGE - k; }
__device__
int m8(int x, int y, int k) { return x + k + EDGE * (y - k); }


__device__
bool isLegal(int x, int y) {
    return board[cur(x, y)] == EMPTY;
}

__device__ char next(char player) {
    return player == 'O' ? 'X' : 'O';
}


__device__
void evaluateHalfline(int x, int y, char player, bool (*h)(int x, int y, int k),
                      int (*m)(int x, int y, int k), int& lin, int& lin1, bool& gap, bool& blocked) {

    for (int i = 1; i < 5; i++) {
        if (h(x, y, i)) {
            if(board[m(x, y, i)] == player) {
                if(!gap)
                    lin++;
                else
                    lin1++;
            }
            else if(board[m(x, y, i)] == EMPTY) {
                if(gap)
                    break;
                else {
                    gap = true;
                }
            }

            else {
                if(!gap || lin1 != 0) {
                    blocked = true;
                }
                else {
                    gap = false;
                }
                break;
            }


        }
        else {
            if(!gap || lin1 != 0) {
                blocked = true;
            }
            else {
                gap = false;
            }
            break;
        }
    }

}


__device__ int payoffs[] = {0, 1, 10, 100, 10000, 1000000, 100000000};

__device__
int getOuts(int lin, int lin1, bool gap, int blocked1, int blocked2) {
    // 5 in a row
    if (lin >= 5)
        return WIN;
    // 4 in a row
    if(lin == 4 && !blocked1 && !blocked2)
        return WINNEXT;
    // 4 blocked from side / 5 with a gap
    if(lin == 4 || lin + lin1 == 4)
        return WINCOND;
    // 3 in a row / 4 with a gap
    if(lin + lin1 == 3 && !blocked1 && !blocked2)
        return WINNEXTCOND;
    // 4 with a gap blocked from side / 2 in a row / 3 blocked from side
    if(lin + lin1 == 3 || (lin == 2 && !blocked1 && !blocked2))
        return POSITIVE;
    // nothing
    return NOTHING;
}

__device__
int reverse(int k) {
    int l = 0;
    for (int i = 0; i < DEPTH; i++) {
        l *= EDGE2;
        l += k % EDGE2;
        k /= EDGE2;
    }
    return l;
}


__device__
bool isFiveInLine(int x, int y, bool (*h1)(int x, int y, int k), bool (*h2)(int x, int y, int k),
                  int (*m1)(int x, int y, int k), int (*m2)(int x, int y, int k)) {
    if(board[cur(x, y)] != EMPTY) {
        char player = board[cur(x, y)];
        int len = 1;
        for (int i = 1; h1(x, y, i) && len < 5; i++) {
            if(board[m1(x, y, i)] == player)
                len++;
            else
                break;
        }
        for (int i = 1; h2(x, y, i) && len < 5; i++) {
            if(board[m2(x, y, i)] == player)
                len++;
            else
                break;
        }
        return len == 5;

    }
    return false;
}


__device__
bool isFive(int x, int y) {
    return isFiveInLine(x, y, h1, h2, m1, m2) ||
           isFiveInLine(x, y, h3, h4, m3, m4) ||
           isFiveInLine(x, y, h5, h6, m5, m6) ||
           isFiveInLine(x, y, h7, h8, m7, m8);
}


__device__
bool spaceLeft(int x, int y, char opponent, bool (*h1)(int x, int y, int k), bool (*h2)(int x, int y, int k), int (*m1)(int x, int y, int k), int (*m2)(int x, int y, int k)) {
    int len = 1;
    for (int i = 1; h1(x, y, i) && len < 5; i++) {
        if(board[m1(x, y, i)] != opponent)
            len++;
        else
            break;
    }
    for (int i = 1; h2(x, y, i) && len < 5; i++) {
        if(board[m2(x, y, i)] != opponent)
            len++;
        else
            break;
    }
    return len == 5;
}

__device__
void evaluateLine(int* outs, int x, int y, char player, bool (*h1)(int x, int y, int k), bool (*h2)(int x, int y, int k),
                  int (*m1)(int x, int y, int k), int (*m2)(int x, int y, int k)) {

    bool gap1 = false, gap2, blocked1 = false, blocked2 = false;
    int lin = 1, lin1 = 0, lin2 = 0;
    if (!spaceLeft(x, y, next(player), h1, h2, m1, m2))
        return;

    evaluateHalfline(x, y, player, h1, m1, lin, lin1, gap1, blocked1);
    evaluateHalfline(x, y, player, h2, m2, lin, lin2, gap2, blocked2);
    if (gap1 && gap2) {
        outs[getOuts(lin, lin1, gap1, blocked1, false)]++;
        outs[getOuts(lin, lin2, gap2, blocked2, false)]++;
    }
    else if (gap1)
        outs[getOuts(lin, lin1, gap1, blocked1, blocked2)]++;
    else if (gap2)
        outs[getOuts(lin, lin2, gap2, blocked2, blocked1)]++;
    else
        outs[getOuts(lin, 0, false, blocked1, blocked2)]++;
}

__device__
int evaluateMove(int x, int y, char player) {
    int outs[N_OUTS];
    for(int i = 0; i < N_OUTS; i++)
        outs[i] = 0;
    evaluateLine(outs, x, y, player, h1, h2, m1, m2);
    evaluateLine(outs, x, y, player, h3, h4, m3, m4);
    evaluateLine(outs, x, y, player, h5, h6, m5, m6);
    evaluateLine(outs, x, y, player, h7, h8, m7, m8);
    outs[WININTWO] += outs[WINNEXTCOND] / 2;
    outs[WINNEXTCOND] %= 2;

    if(outs[WINNEXTCOND] > 0 && outs[WINCOND] > 0) {
        outs[WININTWO]++;
        outs[WINNEXTCOND]--;
        outs[WINCOND]--;
    }

    outs[WININTWO] += outs[WINCOND] / 2;
    outs[WINCOND] %= 2;

    int payoff = 0;
    for (int i = 0; i < N_OUTS; i++)
        payoff += outs[i] * payoffs[i];
    return payoff;
}

__global__
void cuMax(int* vals, int* max) {
    __shared__ int t[SZ2];
    int thid = threadIdx.x, pos = thid + SZ * blockIdx.x;

    if (thid >= SZ) {
        t[thid] = -ILLEGAL_MOVE;
        return;
    }
    else
        t[thid] = vals[pos];

    __syncthreads();
    for (int s = SZ2 / 2; s > 0; s /= 2) {
        if (thid < s) {
            if((t[thid + s] > t[thid] && t[thid + s] != ILLEGAL_MOVE) || t[thid] == ILLEGAL_MOVE)
                t[thid] = t[thid + s];
        }
        else
            return;
        __syncthreads();
    }
    if (thid == 0)
        max[blockIdx.x] = t[0];
}

__global__
void cuMin(int* vals, int* min) {
    __shared__ int t[SZ2];
    int thid = threadIdx.x, pos = thid + SZ * blockIdx.x;

    if (thid >= SZ) {
        t[thid] = ILLEGAL_MOVE;
        return;
    }
    else
        t[thid] = vals[pos];

    __syncthreads();
    for (int s = SZ2 / 2; s > 0; s /= 2) {
        if (thid < s) {
            if((t[thid + s] < t[thid] && t[thid + s] != -ILLEGAL_MOVE) || t[thid] == -ILLEGAL_MOVE)
                t[thid] = t[thid + s];
        }
        else
            return;
        __syncthreads();
    }
    if (thid == 0)
        min[blockIdx.x] = t[0];
}

__global__
void evaluate(char* t, int* vals, char startingPlayer, int length, int firstMove) {
    int x, y, mul = 1, rem = blockIdx.x * blockDim.x + firstMove;
    char player;
    board[threadIdx.x] = t[threadIdx.x];
    __syncthreads();

    if (threadIdx.x == 0) {
        int tmp = reverse(rem);
        player = startingPlayer;
        sum = 0;
        int k = 0;
        for (int i = 0; i < DEPTH; i++) {
            x = (tmp % EDGE2) % EDGE, y = (tmp % EDGE2) / EDGE;
            if (isLegal(x, y)) {
                board[tmp % EDGE2] = player;
                if (isFive(x, y)) {
                    vals[rem] = mul * (WINNING_MOVE - k);
                    sum = -1;
                    break;
                }

            } else {
                vals[rem] = ILLEGAL_MOVE;
                sum = -1;
                break;
            }
            player = next(player);
            tmp /= EDGE2;
            mul *= -1;
            k++;
        }
    }
    __syncthreads();
    if(sum == -1) return;

    player = (DEPTH % 2 == 0) ? startingPlayer : next(startingPlayer);

    x = threadIdx.x % EDGE, y = threadIdx.x / EDGE;
    if(isLegal(x, y)) {
        atomicAdd(&sum, PREFER_FIRST * evaluateMove(x, y, player));
        atomicAdd(&sum, -evaluateMove(x, y, next(player)));
    }
    __syncthreads();

    if(threadIdx.x == 0) {
        vals[rem] = sum * mul;
    }
}

}

